#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <iostream>

#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define CFLOAT4(value) (reinterpret_cast<const float4 *>(&(value))[0])

__global__ void kernel(const float *__restrict__ A_ptr,
                       const float *__restrict__ B_ptr,
                       float *__restrict__ result_ptr,
                       const int num_elements) {
  // TODO: Implement the Kernel Logic
    int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
        if (idx < num_elements) {
            auto a = CFLOAT4(A_ptr[idx]);
            auto b = CFLOAT4(B_ptr[idx]);
            float4 c;
            c.x = a.x + b.x;
            c.y = a.y + b.y;
            c.z = a.z + b.z;
            c.w = a.w + b.w;
            FLOAT4(result_ptr[idx]) = c;
        }
  // Sum up all the elements in the input tensor
}

static void launch_kernel(const void *A_ptr, const void *B_ptr, void *output_ptr,
                          const int num_elements) {
  // TODO: Implement the LaunchKernel Logic
  dim3 block_size(num_elements<512 ? num_elements/4 : 128);     
  dim3 grid_size((num_elements + block_size.x*4 - 1) / (block_size.x * 4));
  kernel<<<grid_size, block_size>>>(
    (const float*)A_ptr, (const float*)B_ptr, (float*)output_ptr, num_elements
  );
  //std::cout << "\n\nWarning: Need to implement the this!!\n\n" << std::endl;
  hipDeviceSynchronize();
}

torch::Tensor test_kernel(const torch::Tensor &A, const torch::Tensor &B) {
  torch::Tensor result_tensor = torch::empty_like(A);
  const int element_count = A.numel();

  launch_kernel(A.data_ptr<float>(), B.data_ptr<float>(), result_tensor.data_ptr<float>(),
                element_count);

  return result_tensor;
}

PYBIND11_MODULE(CUDA_Test, m) {
  m.def("test_kernel", &test_kernel, "Test kernel");
}